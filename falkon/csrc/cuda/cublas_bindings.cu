#include "cublas_bindings.h"
#include "utils.cuh"

#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/HIPContext.h>



void cublas_2d_copy_to_dev_async (const int rows, const int cols, const int elemSize, const torch::Tensor& host_tensor, const int lda, torch::Tensor& dev_tensor, const int ldb, const at::cuda::CUDAStream &stream) {
    TORCH_CUDABLAS_CHECK(hipblasSetMatrixAsync(
        rows, cols, elemSize,
        host_tensor.data_ptr(),
        lda,
        dev_tensor.data_ptr(),
        ldb,
        stream.stream()
    ));
}

void cublas_2d_copy_to_dev (const int rows, const int cols, const int elemSize, const torch::Tensor& host_tensor, const int lda, torch::Tensor& dev_tensor, const int ldb) {
    TORCH_CUDABLAS_CHECK(hipblasSetMatrix(
        rows, cols, elemSize,
        host_tensor.data_ptr(),
        lda,
        dev_tensor.data_ptr(),
        ldb
    ));
}

void cublas_2d_copy_to_host_async(const int rows, const int cols, const int elemSize, const torch::Tensor& dev_tensor, const int lda, torch::Tensor& host_tensor, const int ldb, const at::cuda::CUDAStream &stream) {
    TORCH_CUDABLAS_CHECK(hipblasGetMatrixAsync(
        rows, cols, elemSize,
        dev_tensor.data_ptr(),
        lda,
        host_tensor.data_ptr(),
        ldb,
        stream.stream()
    ));
}

void cublas_2d_copy_to_host(const int rows, const int cols, const int elemSize, const torch::Tensor& dev_tensor, const int lda, torch::Tensor& host_tensor, const int ldb) {
    TORCH_CUDABLAS_CHECK(hipblasGetMatrix(
        rows, cols, elemSize,
        dev_tensor.data_ptr(),
        lda,
        host_tensor.data_ptr(),
        ldb
    ));
}


/* TRSM (cuBLAS) */
template<typename scalar_t>
void trsm(hipblasHandle_t cublas_handle,
                 hipblasSideMode_t side,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 hipblasDiagType_t diag,
                 int m,
                 int n,
                 const scalar_t *alpha,
                 const scalar_t *A,
                 int lda,
                 scalar_t *B,
                 int ldb)
{ throw std::invalid_argument("scalar_t"); }
template<>
void trsm<double>(
                 hipblasHandle_t cublas_handle,
                 hipblasSideMode_t side,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 hipblasDiagType_t diag,
                 int m,
                 int n,
                 const double *alpha,
                 const double *A,
                 int lda,
                 double *B,
                 int ldb)
{
    TORCH_CUDABLAS_CHECK(hipblasDtrsm(cublas_handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb));
}
template<>
void trsm<float>(
                 hipblasHandle_t cublas_handle,
                 hipblasSideMode_t side,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 hipblasDiagType_t diag,
                 int m,
                 int n,
                 const float *alpha,
                 const float *A,
                 int lda,
                 float *B,
                 int ldb)
{
    TORCH_CUDABLAS_CHECK(hipblasStrsm(cublas_handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb));
}

void cublas_trsm(const torch::Tensor& A, const torch::Tensor& B, torch::Scalar alpha, bool left, bool upper, bool transpose, bool unitriangular, int m, int n, int lda, int ldb) {
    hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
    hipblasDiagType_t diag = unitriangular ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT;
    hipblasOperation_t trans = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasSideMode_t side = left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "cublas_trsm", [&]{
        auto handle = at::cuda::getCurrentCUDABlasHandle();
        auto A_data = A.data_ptr<scalar_t>();
        auto B_data = B.data_ptr<scalar_t>();
        scalar_t cast_alpha = alpha.to<scalar_t>();
        trsm<scalar_t>(handle, side, uplo, trans, diag, m, n, &cast_alpha, A_data, lda, B_data, ldb);
    });
}

/* TRMM */
template<typename scalar_t>
void trmm(hipblasHandle_t cublas_handle,
                 hipblasSideMode_t side,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 hipblasDiagType_t diag,
                 int m,
                 int n,
                 const scalar_t *alpha,
                 const scalar_t *A,
                 int lda,
                 scalar_t *B,
                 int ldb,
                 scalar_t *C,
                 int ldc)
{ throw std::invalid_argument("scalar_t"); }
template<>
void trmm<double>(
                 hipblasHandle_t cublas_handle,
                 hipblasSideMode_t side,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 hipblasDiagType_t diag,
                 int m,
                 int n,
                 const double *alpha,
                 const double *A,
                 int lda,
                 double *B,
                 int ldb,
                 double *C,
                 int ldc)
{
    TORCH_CUDABLAS_CHECK(hipblasDtrmm(cublas_handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb, C, ldc));
}
template<>
void trmm<float>(
                 hipblasHandle_t cublas_handle,
                 hipblasSideMode_t side,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 hipblasDiagType_t diag,
                 int m,
                 int n,
                 const float *alpha,
                 const float *A,
                 int lda,
                 float *B,
                 int ldb,
                 float *C,
                 int ldc)
{
    TORCH_CUDABLAS_CHECK(hipblasStrmm(cublas_handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb, C, ldc));
}

void cublas_trmm(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, bool left, bool upper, bool transpose, bool unitriangular, torch::Scalar alpha, int m, int n, int lda, int ldb, int ldc) {
    hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
    hipblasDiagType_t diag = unitriangular ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT;
    hipblasOperation_t trans = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasSideMode_t side = left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "cublas_trmm", [&]{
        auto handle = at::cuda::getCurrentCUDABlasHandle();
        auto A_data = A.data_ptr<scalar_t>();
        auto B_data = B.data_ptr<scalar_t>();
        auto C_data = C.data_ptr<scalar_t>();
        scalar_t cast_alpha = alpha.to<scalar_t>();
        trmm<scalar_t>(handle, side, uplo, trans, diag, m, n, &cast_alpha, A_data, lda, B_data, ldb, C_data, ldc);
    });
}

/* GEMM */

template<typename scalar_t>
void gemm(hipblasHandle_t cublas_handle,
                 hipblasOperation_t transa,
                 hipblasOperation_t transb,
                 int m,
                 int n,
                 int k,
                 const scalar_t *alpha,
                 const scalar_t *A,
                 int lda,
                 const scalar_t *B,
                 int ldb,
                 const scalar_t *beta,
                 scalar_t *C,
                 int ldc)
{ throw std::invalid_argument("scalar_t"); }
template<>
void gemm<double>(
                 hipblasHandle_t cublas_handle,
                 hipblasOperation_t transa,
                 hipblasOperation_t transb,
                 int m,
                 int n,
                 int k,
                 const double *alpha,
                 const double *A,
                 int lda,
                 const double *B,
                 int ldb,
                 const double *beta,
                 double *C,
                 int ldc)
{
    TORCH_CUDABLAS_CHECK(hipblasDgemm(cublas_handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc));
}
template<>
void gemm<float>(
                 hipblasHandle_t cublas_handle,
                 hipblasOperation_t transa,
                 hipblasOperation_t transb,
                 int m,
                 int n,
                 int k,
                 const float *alpha,
                 const float *A,
                 int lda,
                 const float *B,
                 int ldb,
                 const float *beta,
                 float *C,
                 int ldc)
{
    TORCH_CUDABLAS_CHECK(hipblasSgemm(cublas_handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc));
}

void cublas_gemm(const torch::Tensor& A, int lda, bool transa, const torch::Tensor& B, int ldb, bool transb, const torch::Tensor& C, int ldc, int m, int n, int k, torch::Scalar alpha, torch::Scalar beta) {
    hipblasOperation_t transa_op = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb_op = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "cublas_gemm", [&]{
        auto handle = at::cuda::getCurrentCUDABlasHandle();
        auto A_data = A.data_ptr<scalar_t>();
        auto B_data = B.data_ptr<scalar_t>();
        auto C_data = C.data_ptr<scalar_t>();
        scalar_t cast_alpha = alpha.to<scalar_t>();
        scalar_t cast_beta = beta.to<scalar_t>();

        gemm<scalar_t>(handle, transa_op, transb_op, m, n, k, &cast_alpha, A_data, lda, B_data, ldb, &cast_beta, C_data, ldc);
    });
}


/* SYRK */

template<typename scalar_t>
void syrk(hipblasHandle_t cublas_handle,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 int n,
                 int k,
                 const scalar_t *alpha,
                 const scalar_t *A,
                 int lda,
                 const scalar_t *beta,
                 scalar_t *C,
                 int ldc)
{ throw std::invalid_argument("scalar_t"); }
template<>
void syrk<double>(
                 hipblasHandle_t cublas_handle,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 int n,
                 int k,
                 const double *alpha,
                 const double *A,
                 int lda,
                 const double *beta,
                 double *C,
                 int ldc)
{
    TORCH_CUDABLAS_CHECK(hipblasDsyrk(cublas_handle, uplo, trans, n, k, alpha, A, lda, beta, C, ldc));
}
template<>
void syrk<float>(
                 hipblasHandle_t cublas_handle,
                 hipblasFillMode_t uplo,
                 hipblasOperation_t trans,
                 int n,
                 int k,
                 const float *alpha,
                 const float *A,
                 int lda,
                 const float *beta,
                 float *C,
                 int ldc)
{
    TORCH_CUDABLAS_CHECK(hipblasSsyrk(cublas_handle, uplo, trans, n, k, alpha, A, lda, beta, C, ldc));
}

void cublas_syrk(const torch::Tensor& A, int lda, const torch::Tensor& C, int ldc, torch::Scalar alpha, torch::Scalar beta, bool upper, bool transpose, int n, int k) {
    hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t op = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "cublas_syrk", [&]{
        auto handle = at::cuda::getCurrentCUDABlasHandle();
        auto A_data = A.data_ptr<scalar_t>();
        auto C_data = C.data_ptr<scalar_t>();
        scalar_t cast_alpha = alpha.to<scalar_t>();
        scalar_t cast_beta = beta.to<scalar_t>();
        syrk<scalar_t>(handle, uplo, op, n, k, &cast_alpha, A_data, lda, &cast_beta, C_data, ldc);
    });
}
