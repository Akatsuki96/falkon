#include "cublas_bindings.h"
#include "../helpers.h"
#include "cuda_helpers.cuh"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/library.h>

namespace falkon {
namespace ops {


/*
 * TRSM
 */
template<typename scalar_t>
void trsm(
        hipblasHandle_t cublas_handle,
        hipblasSideMode_t side,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        hipblasDiagType_t diag,
        int m,
        int n,
        const scalar_t *alpha,
        const scalar_t *A,
        int lda,
        scalar_t *B,
        int ldb) {
    throw std::invalid_argument("scalar_t");
}
template<>
void trsm<double>(
        hipblasHandle_t cublas_handle,
        hipblasSideMode_t side,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        hipblasDiagType_t diag,
        int m,
        int n,
        const double *alpha,
        const double *A,
        int lda,
        double *B,
        int ldb) {
    FLK_CUDABLAS_CHECK(hipblasDtrsm(cublas_handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb));
}
template<>
void trsm<float>(
        hipblasHandle_t cublas_handle,
        hipblasSideMode_t side,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        hipblasDiagType_t diag,
        int m,
        int n,
        const float *alpha,
        const float *A,
        int lda,
        float *B,
        int ldb) {
    FLK_CUDABLAS_CHECK(hipblasStrsm(cublas_handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb));
}


/*
 * TRMM
 */
template<typename scalar_t>
void trmm(
        hipblasHandle_t cublas_handle,
        hipblasSideMode_t side,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        hipblasDiagType_t diag,
        int m,
        int n,
        const scalar_t *alpha,
        const scalar_t *A,
        int lda,
        const scalar_t *B,
        int ldb,
        scalar_t *C,
        int ldc) {
    throw std::invalid_argument("scalar_t");
}
template<>
void trmm<double>(
        hipblasHandle_t cublas_handle,
        hipblasSideMode_t side,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        hipblasDiagType_t diag,
        int m,
        int n,
        const double *alpha,
        const double *A,
        int lda,
        const double *B,
        int ldb,
        double *C,
        int ldc) {
    FLK_CUDABLAS_CHECK(hipblasDtrmm(cublas_handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb, C, ldc));
}
template<>
void trmm<float>(
        hipblasHandle_t cublas_handle,
        hipblasSideMode_t side,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        hipblasDiagType_t diag,
        int m,
        int n,
        const float *alpha,
        const float *A,
        int lda,
        const float *B,
        int ldb,
        float *C,
        int ldc) {
    FLK_CUDABLAS_CHECK(hipblasStrmm(cublas_handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb, C, ldc));
}


/*
 * GEMM
 */
template<typename scalar_t>
void gemm(
        hipblasHandle_t cublas_handle,
        hipblasOperation_t transa,
        hipblasOperation_t transb,
        int m,
        int n,
        int k,
        const scalar_t *alpha,
        const scalar_t *A,
        int lda,
        const scalar_t *B,
        int ldb,
        const scalar_t *beta,
        scalar_t *C,
        int ldc) {
    throw std::invalid_argument("scalar_t");
}
template<>
void gemm<double>(
        hipblasHandle_t cublas_handle,
        hipblasOperation_t transa,
        hipblasOperation_t transb,
        int m,
        int n,
        int k,
        const double *alpha,
        const double *A,
        int lda,
        const double *B,
        int ldb,
        const double *beta,
        double *C,
        int ldc) {
    FLK_CUDABLAS_CHECK(hipblasDgemm(cublas_handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc));
}
template<>
void gemm<float>(
        hipblasHandle_t cublas_handle,
        hipblasOperation_t transa,
        hipblasOperation_t transb,
        int m,
        int n,
        int k,
        const float *alpha,
        const float *A,
        int lda,
        const float *B,
        int ldb,
        const float *beta,
        float *C,
        int ldc) {
    FLK_CUDABLAS_CHECK(hipblasSgemm(cublas_handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc));
}


/*
 * SYRK
 */
template<typename scalar_t>
void syrk(
        hipblasHandle_t cublas_handle,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        int n,
        int k,
        const scalar_t *alpha,
        const scalar_t *A,
        int lda,
        const scalar_t *beta,
        scalar_t *C,
        int ldc) {
    throw std::invalid_argument("scalar_t");
}
template<>
void syrk<double>(
        hipblasHandle_t cublas_handle,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        int n,
        int k,
        const double *alpha,
        const double *A,
        int lda,
        const double *beta,
        double *C,
        int ldc) {
    FLK_CUDABLAS_CHECK(hipblasDsyrk(cublas_handle, uplo, trans, n, k, alpha, A, lda, beta, C, ldc));
}
template<>
void syrk<float>(
        hipblasHandle_t cublas_handle,
        hipblasFillMode_t uplo,
        hipblasOperation_t trans,
        int n,
        int k,
        const float *alpha,
        const float *A,
        int lda,
        const float *beta,
        float *C,
        int ldc) {
    FLK_CUDABLAS_CHECK(hipblasSsyrk(cublas_handle, uplo, trans, n, k, alpha, A, lda, beta, C, ldc));
}


namespace {

/*
 * Copies
 */
void cublas_2d_copy_to_dev_async (
        const int64_t rows,
        const int64_t cols,
        const int64_t elemSize,
        const at::Tensor& host_tensor,
        const int64_t lda,
        at::Tensor& dev_tensor,
        const int64_t ldb) {
    at::cuda::CUDAStream torch_stream = at::cuda::getCurrentCUDAStream(at::cuda::current_device());
    FLK_CUDABLAS_CHECK(hipblasSetMatrixAsync(
        rows, cols, elemSize,
        host_tensor.data_ptr(),
        lda,
        dev_tensor.data_ptr(),
        ldb,
        torch_stream.stream()
    ));
}

void cublas_2d_copy_to_dev (
        const int64_t rows,
        const int64_t cols,
        const int64_t elemSize,
        const at::Tensor& host_tensor,
        const int64_t lda,
        at::Tensor& dev_tensor,
        const int64_t ldb) {
    FLK_CUDABLAS_CHECK(hipblasSetMatrix(
        rows, cols, elemSize,
        host_tensor.data_ptr(),
        lda,
        dev_tensor.data_ptr(),
        ldb
    ));
}

void cublas_2d_copy_to_host_async(
        const int64_t rows,
        const int64_t cols,
        const int64_t elemSize,
        const at::Tensor& dev_tensor,
        const int64_t lda,
        at::Tensor& host_tensor,
        const int64_t ldb) {
    at::cuda::CUDAStream torch_stream = at::cuda::getCurrentCUDAStream(at::cuda::current_device());
    FLK_CUDABLAS_CHECK(hipblasGetMatrixAsync(
        rows, cols, elemSize,
        dev_tensor.data_ptr(),
        lda,
        host_tensor.data_ptr(),
        ldb,
        torch_stream.stream()
    ));
}

void cublas_2d_copy_to_host(
        const int64_t rows,
        const int64_t cols,
        const int64_t elemSize,
        const at::Tensor& dev_tensor,
        const int64_t lda,
        at::Tensor& host_tensor,
        const int64_t ldb) {
    FLK_CUDABLAS_CHECK(hipblasGetMatrix(
        rows, cols, elemSize,
        dev_tensor.data_ptr(),
        lda,
        host_tensor.data_ptr(),
        ldb
    ));
}

/*
 * Torch wrappers for linalg functions
 */

void cublas_trsm(
        const at::Tensor& A,
        at::Tensor& B,
        const at::Scalar& alpha,
        bool left,
        bool upper,
        bool transpose,
        bool unitriangular,
        int64_t m,
        int64_t n,
        int64_t lda,
        int64_t ldb) {
    hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
    hipblasDiagType_t diag = unitriangular ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT;
    hipblasOperation_t trans = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasSideMode_t side = left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "cublas_trsm", [&]{
        auto handle = at::cuda::getCurrentCUDABlasHandle();
        auto A_data = A.data_ptr<scalar_t>();
        auto B_data = B.data_ptr<scalar_t>();
        scalar_t cast_alpha = alpha.to<scalar_t>();
        trsm<scalar_t>(handle, side, uplo, trans, diag, m, n, &cast_alpha, A_data, lda, B_data, ldb);
    });
}

void cublas_trmm(
        const at::Tensor& A,
        const at::Tensor& B,
        at::Tensor& C,
        bool left,
        bool upper,
        bool transpose,
        bool unitriangular,
        const at::Scalar& alpha,
        int64_t m,
        int64_t n,
        int64_t lda,
        int64_t ldb,
        int64_t ldc) {
    hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
    hipblasDiagType_t diag = unitriangular ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT;
    hipblasOperation_t trans = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasSideMode_t side = left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "cublas_trmm", [&]{
        auto handle = at::cuda::getCurrentCUDABlasHandle();
        auto A_data = A.data_ptr<scalar_t>();
        auto B_data = B.data_ptr<scalar_t>();
        auto C_data = C.data_ptr<scalar_t>();
        scalar_t cast_alpha = alpha.to<scalar_t>();
        trmm<scalar_t>(handle, side, uplo, trans, diag, m, n, &cast_alpha, A_data, lda, B_data, ldb, C_data, ldc);
    });
}

void cublas_gemm(
        const at::Tensor& A,
        int64_t lda,
        bool transa,
        const at::Tensor& B,
        int64_t ldb,
        bool transb,
        at::Tensor& C,
        int64_t ldc,
        int64_t m,
        int64_t n,
        int64_t k,
        const at::Scalar& alpha,
        const at::Scalar& beta) {
    hipblasOperation_t transa_op = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb_op = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "cublas_gemm", [&]{
        auto handle = at::cuda::getCurrentCUDABlasHandle();
        auto A_data = A.data_ptr<scalar_t>();
        auto B_data = B.data_ptr<scalar_t>();
        auto C_data = C.data_ptr<scalar_t>();
        scalar_t cast_alpha = alpha.to<scalar_t>();
        scalar_t cast_beta = beta.to<scalar_t>();

        gemm<scalar_t>(handle, transa_op, transb_op, m, n, k, &cast_alpha, A_data, lda, B_data, ldb, &cast_beta, C_data, ldc);
    });
}

void cublas_syrk(
        const at::Tensor& A,
        int64_t lda,
        at::Tensor& C,
        int64_t ldc,
        const at::Scalar& alpha,
        const at::Scalar& beta,
        bool upper,
        bool transpose,
        int64_t n,
        int64_t k) {
    hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t op = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "cublas_syrk", [&]{
        auto handle = at::cuda::getCurrentCUDABlasHandle();
        auto A_data = A.data_ptr<scalar_t>();
        auto C_data = C.data_ptr<scalar_t>();
        scalar_t cast_alpha = alpha.to<scalar_t>();
        scalar_t cast_beta = beta.to<scalar_t>();
        syrk<scalar_t>(handle, uplo, op, n, k, &cast_alpha, A_data, lda, &cast_beta, C_data, ldc);
    });
}

} // namespace

TORCH_LIBRARY_IMPL(falkon, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("falkon::cublas_2d_copy_to_dev_async"),
      TORCH_FN(cublas_2d_copy_to_dev_async));
  m.impl(
      TORCH_SELECTIVE_NAME("falkon::cublas_2d_copy_to_dev"),
      TORCH_FN(cublas_2d_copy_to_dev));
  m.impl(
      TORCH_SELECTIVE_NAME("falkon::cublas_2d_copy_to_host_async"),
      TORCH_FN(cublas_2d_copy_to_host_async));
  m.impl(
      TORCH_SELECTIVE_NAME("falkon::cublas_2d_copy_to_host"),
      TORCH_FN(cublas_2d_copy_to_host));

  m.impl(
      TORCH_SELECTIVE_NAME("falkon::cublas_trsm"),
      TORCH_FN(cublas_trsm)
  );

  m.impl(
      TORCH_SELECTIVE_NAME("falkon::cublas_trmm"),
      TORCH_FN(cublas_trmm)
  );

  m.impl(
      TORCH_SELECTIVE_NAME("falkon::cublas_gemm"),
      TORCH_FN(cublas_gemm)
  );

  m.impl(
      TORCH_SELECTIVE_NAME("falkon::cublas_syrk"),
      TORCH_FN(cublas_syrk)
  );
}

} // namespace ops
} // namespace falkon
